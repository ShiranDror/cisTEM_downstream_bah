#include "hip/hip_runtime.h"
/*
 * DFTbyDecomposition.cpp
 *
 *  Created on: Oct 21, 2020
 *      Author: himesb
 */

#include "gpu_core_headers.h"
#include "/groups/himesb/cufftdx/include/cufftdx.hpp"
// block_io depends on fp16. Both included from version () on 2020 Oct 30 as IO here may break on future changes
#include "fp16_common.hpp"
#include "block_io.hpp"

using namespace cufftdx;



__global__ void DFT_R2C_WithPaddingKernel(hipfftReal* input_values, hipfftComplex* output_values, int4 dims_in, int4 dims_out, float C);
__global__ void DFT_C2C_WithPaddingKernel_strided(hipfftComplex* input_values, int4 dims_in, int4 dims_out, float C);
__global__ void DFT_R2C_WithPaddingKernel_strided(hipfftReal* input_values, hipfftComplex* output_values, int4 dims_in, int4 dims_out, float C);
__global__ void DFT_C2C_WithPaddingKernel(hipfftComplex* input_values, int4 dims_in, int4 dims_out, float C);
__global__ void DFT_C2C_WithPaddingKernel_rdx2(hipfftComplex* input_values, int4 dims_in, int4 dims_out, float C);

template<class FFT>
__global__ void block_fft_kernel_R2C(typename FFT::input_type* input_values, typename FFT::output_type* output_values, int4 dims_in, int4 dims_out, float twid_constant, int n_sectors);

template<class FFT>
__global__ void block_fft_kernel_R2C_strided(typename FFT::input_type* input_values, typename FFT::output_type* output_values, int4 dims_in, int4 dims_out, float twid_constant, int n_sectors);

template<class FFT>
__global__ void block_fft_kernel_C2C_strided(typename FFT::input_type* input_values, typename FFT::output_type* output_values, int4 dims_in, int4 dims_out, float twid_constant, int n_sectors);

template<class FFT>
//__launch_bounds__(FFT::max_threads_per_block) __global__
__global__ void block_fft_kernel_C2C(typename FFT::input_type* input_values, typename FFT::output_type* output_values, int4 dims_in, int4 dims_out, float twid_constant, int n_sectors);

template<class FFT>
__global__ void block_fft_kernel_R2C_rotate(float* input_values, typename FFT::output_type* output_values, int4 dims_in, int4 dims_out, typename FFT::workspace_type workspace);

template<class FFT>
__global__ void block_fft_kernel_C2C_rotate(typename FFT::input_type* input_values, typename FFT::output_type* output_values, int4 dims_in, int4 dims_out, typename FFT::workspace_type workspace);

template<class FFT>
__global__ void block_fft_kernel_C2R_rotate(typename FFT::input_type* input_values, float* output_values, int4 dims_in, int4 dims_out, typename FFT::workspace_type workspace);


using FFT_256          = decltype(Block() + Size<256>() + Type<fft_type::c2c>() +
                     Precision<float>() + ElementsPerThread<2>() + FFTsPerBlock<1>() + SM<700>());
using FFT_16          = decltype(Block() + Size<16>() + Type<fft_type::c2c>() +
                     Precision<float>() + ElementsPerThread<2>() + FFTsPerBlock<1>() + SM<700>());
using FFT_4096_r2c   = decltype(Block() + Size<4096>() + Type<fft_type::r2c>() +
                     Precision<float>() + ElementsPerThread<8>() + FFTsPerBlock<1>() + SM<700>());
using FFT_4096_c2c   = decltype(Block() + Size<4096>() + Type<fft_type::c2c>() +
                     Precision<float>() + ElementsPerThread<8>() + FFTsPerBlock<1>() + SM<700>());
using FFT_4096_c2r   = decltype(Block() + Size<4096>() + Type<fft_type::c2r>() +
                     Precision<float>() + ElementsPerThread<8>() + FFTsPerBlock<1>() + SM<700>());

DFTbyDecomposition::DFTbyDecomposition() // @suppress("Class members should be properly initialized")
{
	is_set_gpu_images = false;
	is_set_twiddles = false;
	is_allocated_rotated_buffer = false;
//	is_set_outputs = false;
}

DFTbyDecomposition::~DFTbyDecomposition()
{
	if (is_set_twiddles)
	{
		cudaErr(hipFree(twiddles));
	}
	if (is_allocated_rotated_buffer)
	{
		cudaErr(hipFree(d_rotated_buffer));

	}
//	if (is_set_outputs)
//	{
//		cudaErr(hipFree(output_real));
//		cudaErr(hipFree(output_imag));
//	}
}

DFTbyDecomposition::DFTbyDecomposition(const DFTbyDecomposition &other)
{
	// TODO Auto-generated constructor stub

}

DFTbyDecomposition& DFTbyDecomposition::operator=(
		const DFTbyDecomposition &other) {
	// TODO Auto-generated method stub

}

void DFTbyDecomposition::InitTestCase(int wanted_input_size_x, int wanted_input_size_y, int wanted_output_size_x, int wanted_output_size_y)
{
	dims_input = make_int2(wanted_input_size_x, wanted_input_size_y);
	dims_output = make_int2(wanted_output_size_x, wanted_output_size_y);

	// In practice we'll give a pointer to the arrays in some GpuImages
}

void DFTbyDecomposition::SetGpuImages(Image& cpu_input, Image& cpu_output)
{

	// Should be in real space, TODO add check
	input_image.CopyFromCpuImage(cpu_input);
	input_image.CopyHostToDevice();


	// Initialize to Fourier space
	output_image.CopyFromCpuImage(cpu_output);
	output_image.Allocate((int)dims_output.x, (int)dims_output.y, 1, false);
	output_image.Zeros();

	wxPrintf("Sizes in init %d %d in and %d %d out\n",input_image.dims.x, input_image.dims.y, output_image.dims.x, output_image.dims.y);

	is_set_gpu_images = true;

}

void DFTbyDecomposition::AllocateRotatedBuffer()
{
	MyAssertTrue(is_set_gpu_images,"Gpu images must be set before allocating a buffer");

	cudaErr(hipMalloc(&d_rotated_buffer, sizeof(float)*output_image.real_memory_allocated));

	is_allocated_rotated_buffer = true;
}


void DFTbyDecomposition::DFT_R2C_WithPadding()
{

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");



	int threadsPerBlock = input_image.dims.x; // FIXME make sure its a multiple of 32
	int gridDims = input_image.dims.y;
//	dim3 gridDims = dim3((output_image.dims.w/2 + threadsPerBlock - 1) / threadsPerBlock,
//					  	1, 1);
//  output_image.dims.y
	int shared_mem = sizeof(float)*input_image.dims.x;
	float C = -2*PIf/output_image.dims.x;
	DFT_R2C_WithPaddingKernel<< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( input_image.real_values_gpu,  output_image.complex_values_gpu, input_image.dims, output_image.dims, C);
	hipStreamSynchronize(hipStreamPerThread);



}

__global__ void DFT_R2C_WithPaddingKernel(hipfftReal* input_values, hipfftComplex* output_values, int4 dims_in, int4 dims_out, float C)
{

//	// Initialize the shared memory, assuming everying matches the input data X size in
	extern __shared__ float s[];
	// Avoid N*k type conversion and multiplication
	float* data = s;
//	float* coeff= (float*)&data[dims_in.x];


	int x = threadIdx.x;
	int pixel_out = (dims_out.w/2)*blockIdx.x;


	data[x] = __ldg((const float *)&input_values[dims_in.w*blockIdx.x + x]);
	__syncthreads();
//
//	 Loop over N updating the actual twiddle value along the way. This might lead to accuracy problems.
	float sum_real;
	float sum_imag;
	float twi_r;
	float twi_i;
	float coeff;

	for (int k = threadIdx.x; k < dims_out.w/2; k+=blockDim.x)
	{
		coeff = C*(float)k;
		sum_real = 0.0f;
		sum_imag = 0.0f;
		for (int n = 0; n < dims_in.x; n++)
		{
			__sincosf(coeff*n,&twi_i,&twi_r);
			sum_real = __fmaf_rn(data[n],twi_r,sum_real);
			sum_imag = __fmaf_rn(data[n],twi_i,sum_imag);
		}

		// Not sure if an async write, or storage to a shared mem temp would be faster.
		output_values[pixel_out + k].x = sum_real;
		output_values[pixel_out + k].y = sum_imag;
	}


	return;

}


void DFTbyDecomposition::DFT_C2C_WithPadding()
{

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");


	int threadsPerBlock = input_image.dims.x; // FIXME make sure its a multiple of 32
	int gridDims = output_image.dims.w/2;

	int shared_mem = sizeof(hipfftComplex)*input_image.dims.x;

	float C = -2*PIf/output_image.dims.x;
	DFT_C2C_WithPaddingKernel<< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( output_image.complex_values_gpu, input_image.dims, output_image.dims, C);
	hipStreamSynchronize(hipStreamPerThread);



}

__global__ void DFT_C2C_WithPaddingKernel(hipfftComplex* inplace_image, int4 dims_in, int4 dims_out, float C)
{

	// Initialize the shared memory, assuming everying matches the input data X size in
	// Check that setting hipFuncSetSharedMemConfig  to 8byte makes any diff for complex reads
	extern __shared__ hipfftComplex c[];
	hipfftComplex* data = c;


	int x = threadIdx.x;
	int pixel_out = (dims_out.w/2)*blockIdx.x;

	data[x] = __ldg((const hipfftComplex *)&inplace_image[pixel_out + x]);
	__syncthreads();
//
//	 Loop over N updating the actual twiddle value along the way. This might lead to accuracy problems.
	hipfftComplex sum;
	float twi_r;
	float twi_i;
	float coeff;
	float tmp;

	for (int k = threadIdx.x; k < dims_out.w/2; k+=blockDim.x)
	{
		coeff = C*(float)k;
		sum.x = 0.0f;
		sum.y = 0.0f;
		for (int n = 0; n < dims_in.y; n++)
		{
			__sincosf(coeff*n,&twi_i,&twi_r);
			tmp = data[n].x * twi_i;
			sum.x += __fmaf_rn(data[n].x, twi_r, -twi_i * data[n].y);
			sum.y += __fmaf_rn(data[n].y, twi_r, tmp);
		}

		// Not sure if an async write, or storage to a shared mem temp would be faster.
//		inplace_image[pixel_out + k].x = sum_real;
//		inplace_image[pixel_out + k].y = sum_imag;
		inplace_image[pixel_out + k] = sum;
	}



	return;

}


void DFTbyDecomposition::DFT_R2C_WithPadding_strided()
{

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");



	int threadsPerBlock = input_image.dims.y; // FIXME make sure its a multiple of 32
	int gridDims = input_image.dims.x;
//	dim3 gridDims = dim3((output_image.dims.w/2 + threadsPerBlock - 1) / threadsPerBlock,
//					  	1, 1);
//  output_image.dims.y
	int shared_mem = sizeof(float)*input_image.dims.y;
	float C = -2*PIf/output_image.dims.y;
	DFT_R2C_WithPaddingKernel_strided<< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( input_image.real_values_gpu,  output_image.complex_values_gpu, input_image.dims, output_image.dims, C);
	hipStreamSynchronize(hipStreamPerThread);



}

__global__ void DFT_R2C_WithPaddingKernel_strided(hipfftReal* input_values, hipfftComplex* output_values, int4 dims_in, int4 dims_out, float C)
{

//	// Initialize the shared memory, assuming everying matches the input data X size in
	extern __shared__ float s[];
	// Avoid N*k type conversion and multiplication
	float* data = s;
//	float* coeff= (float*)&data[dims_in.x];


	int y = threadIdx.x;
	int pixel_in = blockIdx.x + y * (dims_in.w);

	data[y] = __ldg((const hipfftReal *)&input_values[pixel_in]);
	__syncthreads();
//

//
//	 Loop over N updating the actual twiddle value along the way. This might lead to accuracy problems.
	float sum_real;
	float sum_imag;
	float twi_r;
	float twi_i;
	float coeff;

	for (int k = threadIdx.x; k < dims_out.y; k+=blockDim.x)
	{
		coeff = C*(float)k;
		sum_real = 0.0f;
		sum_imag = 0.0f;
		for (int n = 0; n < dims_in.x; n++)
		{
			__sincosf(coeff*n,&twi_i,&twi_r);
			sum_real = __fmaf_rn(data[n],twi_r,sum_real);
			sum_imag = __fmaf_rn(data[n],twi_i,sum_imag);
		}

		// Not sure if an async write, or storage to a shared mem temp would be faster.
		// Not sure if an async write, or storage to a shared mem temp would be faster.
		output_values[blockIdx.x + k * (dims_out.w/2)].x = sum_real;
		output_values[blockIdx.x + k * (dims_out.w/2)].y = sum_imag;
	}


	return;

}


void DFTbyDecomposition::DFT_C2C_WithPadding_strided()
{

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");


	int threadsPerBlock = input_image.dims.y; // FIXME make sure its a multiple of 32
	int gridDims = output_image.dims.w/2;

	int shared_mem = sizeof(hipfftComplex)*input_image.dims.y;

	float C = -2*PIf/output_image.dims.y;
	DFT_C2C_WithPaddingKernel_strided<< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( output_image.complex_values_gpu, input_image.dims, output_image.dims, C);
	hipStreamSynchronize(hipStreamPerThread);



}

__global__ void DFT_C2C_WithPaddingKernel_strided(hipfftComplex* inplace_image, int4 dims_in, int4 dims_out, float C)
{

	// Initialize the shared memory, assuming everying matches the input data X size in
	// Check that setting hipFuncSetSharedMemConfig  to 8byte makes any diff for complex reads
	extern __shared__ hipfftComplex c[];
	hipfftComplex* data = c;


	int y = threadIdx.x;
	int pixel_in = blockIdx.x + y * (dims_out.w/2);


	data[y] = __ldg((const hipfftComplex *)&inplace_image[pixel_in]);
	__syncthreads();
//
//	 Loop over N updating the actual twiddle value along the way. This might lead to accuracy problems.
	float sum_real;
	float sum_imag;
	float twi_r;
	float twi_i;
	float coeff;
	float tmp;

	for (int k = threadIdx.x; k < dims_out.y; k+=blockDim.x)
	{
		coeff = C*(float)k;
		sum_real = 0.0f;
		sum_imag = 0.0f;
		for (int n = 0; n < dims_in.y; n++)
		{
			__sincosf(coeff*n,&twi_i,&twi_r);
			tmp = data[n].x * twi_i;
			sum_real += __fmaf_rn(data[n].x, twi_r, -twi_i * data[n].y);
			sum_imag += __fmaf_rn(data[n].y, twi_r, tmp);
		}

		// Not sure if an async write, or storage to a shared mem temp would be faster.
		inplace_image[blockIdx.x + k * (dims_out.w/2)].x = sum_real;
		inplace_image[blockIdx.x + k * (dims_out.w/2)].y = sum_imag;
	}


	return;

}

void DFTbyDecomposition::DFT_C2C_WithPadding_rdx2()
{

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");


	int threadsPerBlock = input_image.dims.x; // FIXME make sure its a multiple of 32
	int gridDims = output_image.dims.w/2;

	int shared_mem = sizeof(hipfftComplex)*input_image.dims.x;

	float C = -2*PIf/output_image.dims.x*2;
	DFT_C2C_WithPaddingKernel_rdx2<< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( output_image.complex_values_gpu, input_image.dims, output_image.dims, C);
	hipStreamSynchronize(hipStreamPerThread);



}

__global__ void DFT_C2C_WithPaddingKernel_rdx2(hipfftComplex* inplace_image, int4 dims_in, int4 dims_out, float C)
{

	// Initialize the shared memory, assuming everying matches the input data X size in
	// Check that setting hipFuncSetSharedMemConfig  to 8byte makes any diff for complex reads
	extern __shared__ hipfftComplex c[];
	hipfftComplex* data = c;


	int x = threadIdx.x;
	int pixel_out = (dims_out.w/2)*blockIdx.x;

	data[x] = __ldg((const hipfftComplex *)&inplace_image[pixel_out + x]);
	__syncthreads();
//
//	 Loop over N updating the actual twiddle value along the way. This might lead to accuracy problems.
	hipfftComplex sum;
	hipfftComplex eve;
	float twi_r;
	float twi_i;
	float coeff;
	float tmp;

	for (int k = threadIdx.x; k < dims_out.w/4; k+=blockDim.x)
	{
		// get the even DFT
		coeff = C*(float)k;
		sum.x = 0.0f;
		sum.y = 0.0f;
		for (int n = 0; n < dims_in.y; n+=2)
		{
			__sincosf(coeff*n,&twi_i,&twi_r);
			tmp = data[n].x * twi_i;
			sum.x += __fmaf_rn(data[n].x, twi_r, -twi_i * data[n].y);
			sum.y += __fmaf_rn(data[n].y, twi_r, tmp);
		}

		eve = sum;

		// get the odd DFT
		sum.x = 0.0f;
		sum.y = 0.0f;
		for (int n = 1; n < dims_in.y; n+=2)
		{
			__sincosf(coeff*n,&twi_i,&twi_r);
			tmp = data[n].x * twi_i;
			sum.x += __fmaf_rn(data[n].x, twi_r, -twi_i * data[n].y);
			sum.y += __fmaf_rn(data[n].y, twi_r, tmp);
		}

		// Get the twiddle for the combined radix
		__sincosf(coeff/2.0f,&twi_i,&twi_r);
		// Multiply the odd
		tmp = sum.x * twi_i;
		sum.x = __fmaf_rn(sum.x, twi_r, -twi_i * sum.y);
		sum.y = __fmaf_rn(sum.y, twi_r, tmp);

		inplace_image[pixel_out + k].x = eve.x + sum.x;
		inplace_image[pixel_out + k].y = eve.y + sum.y;

		inplace_image[pixel_out + k + dims_out.w/4].x = eve.x - sum.x;
		inplace_image[pixel_out + k + dims_out.w/4].y = eve.y - sum.y;

	}



	return;

}


void DFTbyDecomposition::FFT_R2C_WithPadding_strided()
{

	// This is the first set of 1d ffts when the input data are real valued, accessing the strided dimension. Since we need the full length, it will actually run a C2C xform

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");

	// Elements per thread must be [2,32]
    const int ept = 2;

    // FFts per block. Might be able to re-use twiddles but prob more mem intensive. TODO test me and also evaluate memory size
    const int ffts_per_block = 1; // 1 is the default.

    // For now consider the simplest launch params, where one input element is handled per thread.
    MyAssertFalse(input_image.dims.y % ept, "The elements per thread is not a divisor of the input y-dimension.");
	int threadsPerBlock = input_image.dims.y / ept; // FIXME make sure its a multiple of 32
	int gridDims = input_image.dims.x;

	// For the twiddle factors ahead of the P size ffts
	float CN = -2*PIf/output_image.dims.y;
	int   IQ = output_image.dims.y / input_image.dims.y; // FIXME assuming for now this is already divisible

    // FFT is defined, its: size, type, direction, precision. Block() operator informs that FFT
    // will be executed on block level. Shared memory is required for co-operation between threads.


	if (input_image.dims.y == 256)
	{
	    using FFT = decltype(FFT_256() + Direction<fft_direction::forward>() );
		int shared_mem = sizeof(FFT::value_type)*(input_image.dims.y) + FFT::shared_memory_size*8;
		block_fft_kernel_R2C_strided<FFT><< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( (typename FFT::input_type*)input_image.real_values_gpu,  (typename FFT::output_type*)output_image.complex_values_gpu, input_image.dims, output_image.dims, CN,IQ);



	}
	else if (input_image.dims.y == 16)
	{
	    using FFT = decltype(FFT_16() + Direction<fft_direction::forward>() );
		int shared_mem = sizeof(float)*(2+input_image.dims.y) + FFT::shared_memory_size*8;
		block_fft_kernel_R2C_strided<FFT><< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( (typename FFT::input_type*)input_image.real_values_gpu,  (typename FFT::output_type*)output_image.complex_values_gpu, input_image.dims, output_image.dims, CN,IQ);


	}
	else
	{
		exit(-1);
	}




}

template<class FFT>
//__launch_bounds__(FFT::max_threads_per_block) __global__
__global__ void block_fft_kernel_R2C_strided(typename FFT::input_type* input_values, typename FFT::output_type* output_values, int4 dims_in, int4 dims_out, float twid_constant, int n_sectors)
{

//	// Initialize the shared memory, assuming everying matches the input data X size in
    using complex_type = typename FFT::value_type;

	extern __shared__  complex_type real_data[];
	complex_type* shared_mem_work  = (complex_type*)&real_data[dims_in.y];
	float* fake_input = reinterpret_cast<float*>(input_values);


	// Memory used by FFT
	complex_type twiddle;
    complex_type thread_data[FFT::storage_size];

    // To re-map the thread index to the data
    int input_MAP[FFT::storage_size];
    // To re-map the decomposed frequency to the full output frequency
    int output_MAP[FFT::storage_size];
    // For a given decomposed fragment
    float twiddle_factors_args[FFT::storage_size];

    // This way reads are
    int i;

    for (i = 0; i < FFT::elements_per_thread; i++)
    {
    	// index into the input data
    	input_MAP[i] = threadIdx.x + i * (size_of<FFT>::value / FFT::elements_per_thread);
		output_MAP[i] = n_sectors * input_MAP[i];
		twiddle_factors_args[i] = twid_constant * input_MAP[i];

		// Unpack the floats and move from shared mem into the register space.in R2C this would happen anyway as a preprocessing step.
		real_data[input_MAP[i]].x = __ldg((const hipfftReal *)&fake_input[blockIdx.x + input_MAP[i] * (dims_in.w)]);
		real_data[input_MAP[i]].y = 0.0f;
    }
	__syncthreads();


	// this data will be re-used for each n_sectors FFTs
    for (i = 0; i < FFT::elements_per_thread; i++)
    {
		thread_data[i] = real_data[input_MAP[i]];
    }


    // For loop zero the twiddles don't need to be computed
    FFT().execute(thread_data, shared_mem_work);

    // The memory access is strided anyway so just send to global
    for (i = 0; i < FFT::elements_per_thread; i++)
    {
        output_values[blockIdx.x + output_MAP[i] * (dims_out.w/2)] = thread_data[i];
    }

    // For the other fragments we need the initial twiddle
	for (int fft_fragment = 1; fft_fragment < n_sectors; fft_fragment++)
	{
	    for (i = 0; i < FFT::elements_per_thread; i++)
	    {
			// Pre shift with twiddle
			__sincosf(twiddle_factors_args[i]*fft_fragment,&twiddle.y,&twiddle.x);
			twiddle *= real_data[input_MAP[i]]; // Only the inplace operators are included in cufftdx::types TODO expand
		    thread_data[i] = twiddle;
	    }

	      FFT().execute(thread_data, shared_mem_work);

		for (i = 0; i < FFT::elements_per_thread; i++)
		{
		      output_values[blockIdx.x + (fft_fragment + output_MAP[i]) * (dims_out.w/2)] = thread_data[i];
		}

	}



	return;

}

void DFTbyDecomposition::FFT_R2C_WithPadding()
{

	// This is the first set of 1d ffts when the input data are real valued, accessing the strided dimension. Since we need the full length, it will actually run a C2C xform

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");

	// Elements per thread must be [2,32]
    const int ept = 2;

    // FFts per block. Might be able to re-use twiddles but prob more mem intensive. TODO test me and also evaluate memory size
    const int ffts_per_block = 1; // 1 is the default.

    // For now consider the simplest launch params, where one input element is handled per thread.
    MyAssertFalse(input_image.dims.x % ept, "The elements per thread is not a divisor of the input y-dimension.");
	int threadsPerBlock = input_image.dims.x / ept; // FIXME make sure its a multiple of 32
	int gridDims = input_image.dims.y;

	// For the twiddle factors ahead of the P size ffts
	float CN = -2*PIf/output_image.dims.x;
	int   IQ = output_image.dims.x / input_image.dims.x; // FIXME assuming for now this is already divisible
    // FFT is defined, its: size, type, direction, precision. Block() operator informs that FFT
    // will be executed on block level. Shared memory is required for co-operation between threads.


	if (input_image.dims.y == 256)
	{
	    using FFT = decltype(FFT_256() + Direction<fft_direction::forward>() );
		int shared_mem = sizeof(FFT::output_type)*(input_image.dims.x) + FFT::shared_memory_size*8;
		block_fft_kernel_R2C<FFT><< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( (typename FFT::input_type*)input_image.real_values_gpu,  (typename FFT::output_type*)output_image.complex_values_gpu, input_image.dims, output_image.dims, CN,IQ);



	}
	else if (input_image.dims.y == 16)
	{
	    using FFT = decltype(FFT_16() + Direction<fft_direction::forward>() );
		int shared_mem = sizeof(FFT::value_type)*(input_image.dims.x) + FFT::shared_memory_size*8;
		block_fft_kernel_R2C<FFT><< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( (typename FFT::input_type*)input_image.real_values_gpu,  (typename FFT::output_type*)output_image.complex_values_gpu, input_image.dims, output_image.dims, CN,IQ);

	}
	else
	{
		exit(-1);
	}




}

template<class FFT>
//__launch_bounds__(FFT::max_threads_per_block) __global__
__global__ void block_fft_kernel_R2C(typename FFT::input_type* input_values, typename FFT::output_type* output_values, int4 dims_in, int4 dims_out, float twid_constant, int n_sectors)
{

//	// Initialize the shared memory, assuming everying matches the input data X size in
    using complex_type = typename FFT::value_type;

	extern __shared__  complex_type shared_mem_work[];
	complex_type* real_data = (complex_type*)&shared_mem_work[FFT::shared_memory_size];
	float* fake_input = reinterpret_cast<float*>(input_values);

	// Memory used by FFT
	complex_type twiddle;
    complex_type thread_data[FFT::storage_size];

    // To re-map the thread index to the data
    int input_MAP[FFT::storage_size];
    // To re-map the decomposed frequency to the full output frequency
    int output_MAP[FFT::storage_size];
    // For a given decomposed fragment
    float twiddle_factors_args[FFT::storage_size];

    // This way reads are
    int i;

    for (i = 0; i < FFT::elements_per_thread; i++)
    {
    	// index into the input data
    	input_MAP[i] = threadIdx.x + i * (size_of<FFT>::value / FFT::elements_per_thread);
		output_MAP[i] = n_sectors * input_MAP[i];
		twiddle_factors_args[i] = twid_constant * input_MAP[i];

		// Unpack the floats and move from shared mem into the register space.in R2C this would happen anyway as a preprocessing step.
		real_data[input_MAP[i]].x = __ldg((const hipfftReal *)&fake_input[blockIdx.x * dims_in.w + input_MAP[i]]);
		real_data[input_MAP[i]].y = 0.0f;
    }
	__syncthreads();


	// this data will be re-used for each n_sectors FFTs
    for (i = 0; i < FFT::elements_per_thread; i++)
    {
		thread_data[i] = real_data[input_MAP[i]];
    }


    // For loop zero the twiddles don't need to be computed
    FFT().execute(thread_data, shared_mem_work);



	// The memory access is strided anyway so just send to global
	for (i = 0; i < FFT::elements_per_thread; i++)
	{
		if (output_MAP[i] < dims_out.w/2) // FIXME we should just do a R2C normal here
		{
			output_values[blockIdx.x  * (dims_out.w/2) + output_MAP[i]] = thread_data[i];
		}
	}


    // For the other fragments we need the initial twiddle
	for (int fft_fragment = 1; fft_fragment < n_sectors; fft_fragment++)
	{
	    for (i = 0; i < FFT::elements_per_thread; i++)
	    {
			// Pre shift with twiddle
			__sincosf(twiddle_factors_args[i]*fft_fragment,&twiddle.y,&twiddle.x);
			twiddle *= real_data[input_MAP[i]]; // Only the inplace operators are included in cufftdx::types TODO expand
		    thread_data[i] = twiddle;
	    }

	      FFT().execute(thread_data, shared_mem_work);

		for (i = 0; i < FFT::elements_per_thread; i++)
		{
		    if ((fft_fragment + output_MAP[i]) < dims_out.w/2) // FIXME we should just do a R2C normal here
		    {
		      output_values[blockIdx.x  * (dims_out.w/2) + (fft_fragment + output_MAP[i])] = thread_data[i];
		    }
		}

	}



	return;

}

void DFTbyDecomposition::FFT_C2C_WithPadding_strided()
{

	// This is the first set of 1d ffts when the input data are real valued, accessing the strided dimension. Since we need the full length, it will actually run a C2C xform

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");

	// Elements per thread must be [2,32]
    const int ept = 2;

    // FFts per block. Might be able to re-use twiddles but prob more mem intensive. TODO test me and also evaluate memory size
    const int ffts_per_block = 1; // 1 is the default.

    // For now consider the simplest launch params, where one input element is handled per thread.
    MyAssertFalse(input_image.dims.y % ept, "The elements per thread is not a divisor of the input y-dimension.");
	int threadsPerBlock = input_image.dims.y / ept; // FIXME make sure its a multiple of 32
	int gridDims = output_image.dims.w/2;

	// For the twiddle factors ahead of the P size ffts
	float CN = -2*PIf/output_image.dims.y;
	int   IQ = output_image.dims.y / input_image.dims.y; // FIXME assuming for now this is already divisible

    // FFT is defined, its: size, type, direction, precision. Block() operator informs that FFT
    // will be executed on block level. Shared memory is required for co-operation between threads.


	if (input_image.dims.y == 256)
	{
	    using FFT = decltype(FFT_256() + Direction<fft_direction::forward>() );
		int shared_mem = sizeof(FFT::value_type)*(input_image.dims.y) + FFT::shared_memory_size*8;
		block_fft_kernel_C2C_strided<FFT><< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( (typename FFT::input_type*)output_image.complex_values_gpu,  (typename FFT::output_type*)output_image.complex_values_gpu, input_image.dims, output_image.dims, CN,IQ);


	}
	else if (input_image.dims.y == 16)
	{
	    using FFT = decltype(FFT_16() + Direction<fft_direction::forward>() );
		int shared_mem = sizeof(float)*(input_image.dims.y) + FFT::shared_memory_size*8;
		block_fft_kernel_C2C_strided<FFT><< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( (typename FFT::input_type*)output_image.complex_values_gpu,  (typename FFT::output_type*)output_image.complex_values_gpu, input_image.dims, output_image.dims, CN,IQ);
		output_image.printVal("out 1",0);
		output_image.printVal("out 1",1);

		output_image.printVal("out 1",2);
		output_image.printVal("out 1",3);
		output_image.printVal("out 1",4);
		output_image.printVal("out 1",5);



	}
	else
	{
		exit(-1);
	}




}

template<class FFT>
//__launch_bounds__(FFT::max_threads_per_block) __global__
__global__ void block_fft_kernel_C2C_strided(typename FFT::input_type* input_values, typename FFT::output_type* output_values, int4 dims_in, int4 dims_out, float twid_constant, int n_sectors)
{

//	// Initialize the shared memory, assuming everying matches the input data X size in
    using complex_type = typename FFT::value_type;

	extern __shared__  complex_type real_data[];
	complex_type* shared_mem_work_3= (complex_type*)&real_data[dims_in.y];


	// Memory used by FFT
	complex_type twiddle;
    complex_type thread_data[FFT::elements_per_thread];

    // To re-map the thread index to the data
    int input_MAP[FFT::elements_per_thread];
    // To re-map the decomposed frequency to the full output frequency
    int output_MAP[FFT::elements_per_thread];
    // For a given decomposed fragment
    float twiddle_factors_args[FFT::elements_per_thread];

    // This way reads are
    int i;

    for (i = 0; i < FFT::elements_per_thread; i++)
    {
    	// index into the input data
    	input_MAP[i] = threadIdx.x + i * (size_of<FFT>::value / FFT::elements_per_thread);
		output_MAP[i] = n_sectors * input_MAP[i];
		twiddle_factors_args[i] = twid_constant * input_MAP[i];

		// Unpack the floats and move from shared mem into the register space.in R2C this would happen anyway as a preprocessing step.
		real_data[input_MAP[i]].x = __ldg((const float*)&input_values[blockIdx.x + input_MAP[i] * (dims_out.w/2)].x);
		real_data[input_MAP[i]].y = __ldg((const float*)&input_values[blockIdx.x + input_MAP[i] * (dims_out.w/2)].y);

    }
	__syncthreads();


	// this data will be re-used for each n_sectors FFTs
    for (i = 0; i < FFT::elements_per_thread; i++)
    {
		thread_data[i] = real_data[input_MAP[i]];
    }


    // For loop zero the twiddles don't need to be computed
    FFT().execute(thread_data, shared_mem_work_3);

    // The memory access is strided anyway so just send to global
    for (i = 0; i < FFT::elements_per_thread; i++)
    {
        output_values[blockIdx.x + output_MAP[i] * (dims_out.w/2)] = thread_data[i];
    }

    // For the other fragments we need the initial twiddle
	for (int fft_fragment = 1; fft_fragment < n_sectors; fft_fragment++)
	{
	    for (i = 0; i < FFT::elements_per_thread; i++)
	    {
			// Pre shift with twiddle
			__sincosf(twiddle_factors_args[i]*fft_fragment,&twiddle.y,&twiddle.x);
			twiddle *= real_data[input_MAP[i]]; // Only the inplace operators are included in cufftdx::types TODO expand
		    thread_data[i] = twiddle;
	    }

	      FFT().execute(thread_data, shared_mem_work_3);

		for (i = 0; i < FFT::elements_per_thread; i++)
		{
		      output_values[blockIdx.x + (fft_fragment + output_MAP[i]) * (dims_out.w/2)] = thread_data[i];
		}

	}



	return;

}

void DFTbyDecomposition::FFT_C2C_WithPadding()
{

	// This is the first set of 1d ffts when the input data are real valued, accessing the strided dimension. Since we need the full length, it will actually run a C2C xform

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");

	// Elements per thread must be [2,32]
    const int ept = 2;

    // FFts per block. Might be able to re-use twiddles but prob more mem intensive. TODO test me and also evaluate memory size
    const int ffts_per_block = 1; // 1 is the default.

    // For now consider the simplest launch params, where one input element is handled per thread.
    MyAssertFalse(input_image.dims.x % ept, "The elements per thread is not a divisor of the input y-dimension.");
	int threadsPerBlock = input_image.dims.x / ept; // FIXME make sure its a multiple of 32
	int gridDims = output_image.dims.y;

	// For the twiddle factors ahead of the P size ffts
	float CN = -2*PIf/output_image.dims.x;
	int   IQ = output_image.dims.x / input_image.dims.x; // FIXME assuming for now this is already divisible
    // FFT is defined, its: size, type, direction, precision. Block() operator informs that FFT
    // will be executed on block level. Shared memory is required for co-operation between threads.


	if (input_image.dims.y == 256)
	{
	    using FFT = decltype(FFT_256() + Direction<fft_direction::forward>() );
		int shared_mem = sizeof(FFT::output_type)*(input_image.dims.x) + FFT::shared_memory_size*8;
		block_fft_kernel_C2C<FFT><< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( (typename FFT::input_type*)input_image.real_values_gpu,  (typename FFT::output_type*)output_image.complex_values_gpu, input_image.dims, output_image.dims, CN,IQ);



	}
	else if (input_image.dims.y == 16)
	{
	    using FFT = decltype(FFT_16() + Direction<fft_direction::forward>() );
		int shared_mem = sizeof(FFT::value_type)*(input_image.dims.x) + FFT::shared_memory_size*8;
		block_fft_kernel_R2C<FFT><< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( (typename FFT::input_type*)input_image.real_values_gpu,  (typename FFT::output_type*)output_image.complex_values_gpu, input_image.dims, output_image.dims, CN,IQ);

	}
	else
	{
		exit(-1);
	}




}

template<class FFT>
//__launch_bounds__(FFT::max_threads_per_block) __global__
__global__ void block_fft_kernel_C2C(typename FFT::input_type* input_values, typename FFT::output_type* output_values, int4 dims_in, int4 dims_out, float twid_constant, int n_sectors)
{

//	// Initialize the shared memory, assuming everying matches the input data X size in
    using complex_type = typename FFT::value_type;

	extern __shared__  complex_type real_data[];
	complex_type* shared_mem_work = (complex_type*)&real_data[dims_in.x];

	// Memory used by FFT
	complex_type twiddle;
    complex_type thread_data[FFT::storage_size];

    // To re-map the thread index to the data
    int input_MAP[FFT::storage_size];

    // To re-map the decomposed frequency to the full output frequency
    int output_MAP[FFT::storage_size];
    // For a given decomposed fragment
    float twiddle_factors_args[FFT::storage_size];

    // This way reads are
    int i;

    for (i = 0; i < FFT::elements_per_thread; i++)
    {
    	// index into the input data
    	input_MAP[i] = threadIdx.x + i * (size_of<FFT>::value / FFT::elements_per_thread);
		output_MAP[i] = n_sectors * input_MAP[i];
		twiddle_factors_args[i] = twid_constant * input_MAP[i];

		// Unpack the floats and move from shared mem into the register space.in R2C this would happen anyway as a preprocessing step.
		real_data[input_MAP[i]].x = __ldg((const hipfftReal *)&output_values[blockIdx.x * dims_in.w/2 + input_MAP[i]].x);
		real_data[input_MAP[i]].y = __ldg((const hipfftReal *)&output_values[blockIdx.x * dims_in.w/2 + input_MAP[i]].y);
    }
	__syncthreads();


	// this data will be re-used for each n_sectors FFTs
    for (i = 0; i < FFT::elements_per_thread; i++)
    {
		thread_data[i] = real_data[input_MAP[i]];
    }


    // For loop zero the twiddles don't need to be computed
    FFT().execute(thread_data, shared_mem_work);



	// The memory access is strided anyway so just send to global
	for (i = 0; i < FFT::elements_per_thread; i++)
	{
		if (output_MAP[i] < dims_out.w/2) // FIXME we should just do a R2C normal here
		{
			output_values[blockIdx.x  * (dims_out.w/2) + output_MAP[i]] = thread_data[i];
		}
	}


    // For the other fragments we need the initial twiddle
	for (int fft_fragment = 1; fft_fragment < n_sectors; fft_fragment++)
	{
	    for (i = 0; i < FFT::elements_per_thread; i++)
	    {
			// Pre shift with twiddle
			__sincosf(twiddle_factors_args[i]*fft_fragment,&twiddle.y,&twiddle.x);
			twiddle *= real_data[input_MAP[i]]; // Only the inplace operators are included in cufftdx::types TODO expand
		    thread_data[i] = twiddle;
	    }

	      FFT().execute(thread_data, shared_mem_work);

		for (i = 0; i < FFT::elements_per_thread; i++)
		{
		    if ((fft_fragment + output_MAP[i]) < dims_out.w/2) // FIXME we should just do a R2C normal here
		    {
		      output_values[blockIdx.x  * (dims_out.w/2) + (fft_fragment + output_MAP[i])] = thread_data[i];
		    }
		}

	}



	return;

}

void DFTbyDecomposition::FFT_R2C_rotate()
{

	// This is the first set of 1d ffts when the input data are real valued, accessing the strided dimension. Since we need the full length, it will actually run a C2C xform

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");

	// Elements per thread must be [2,32]
    const int ept = 8;

    // FFts per block. Might be able to re-use twiddles but prob more mem intensive. TODO test me and also evaluate memory size
    const int ffts_per_block = 1; // 1 is the default.

	int threadsPerBlock = 4096/ept; // FIXME make sure its a multiple of 32
	int gridDims = max(input_image.dims.y, input_image.dims.x);

if (input_image.dims.y == 4096)
	{
	    using FFT = decltype(FFT_4096_r2c() + Direction<fft_direction::forward>() );
		int shared_mem = sizeof(FFT::value_type)*(input_image.dims.x) + FFT::shared_memory_size*8;

	    hipError_t error_code = hipSuccess;
	    auto workspace = make_workspace<FFT>(error_code);
		block_fft_kernel_R2C_rotate<FFT><< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( (float *)input_image.real_values_gpu,  (typename FFT::output_type*)d_rotated_buffer, input_image.dims, output_image.dims, workspace);

	}
	else
	{
		exit(-1);
	}




}

template<class FFT>
__global__ void block_fft_kernel_R2C_rotate(float* input_values, typename FFT::output_type* output_values, int4 dims_in, int4 dims_out, typename FFT::workspace_type workspace)
{

	if (blockIdx.x > dims_in.y) return;

//	// Initialize the shared memory, assuming everyting matches the input data X size in
    using complex_type = typename FFT::value_type;
    using scalar_type  = typename complex_type::value_type;

	extern __shared__  complex_type shared_mem[];

    complex_type thread_data[FFT::storage_size];
    int source_idx[FFT::storage_size];

    bah_io::io<FFT>::load_r2c(&input_values[blockIdx.x * dims_in.w], thread_data, 1, source_idx);

    // For loop zero the twiddles don't need to be computed
    FFT().execute(thread_data, shared_mem);


	// The memory access is strided anyway so just send to global.
    int rotated_offset[2] = { -dims_out.y, blockIdx.x  + (dims_out.y) * (dims_out.w/2 - 1)};
    bah_io::io<FFT>::store_r2c_rotated(thread_data, output_values, 1, rotated_offset);


	return;

}

void DFTbyDecomposition::FFT_C2C_rotate(bool forward_transform)
{

	// This is the first set of 1d ffts when the input data are real valued, accessing the strided dimension. Since we need the full length, it will actually run a C2C xform

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");

	// Elements per thread must be [2,32]
    const int ept = 8;


	int threadsPerBlock = 4096/ept ; // FIXME make sure its a multiple of 32
	int gridDims = output_image.dims.w/2;


	if (input_image.dims.y == 4096)
	{
		if (forward_transform)
		{
		    using FFT = decltype(FFT_4096_c2c() + Direction<fft_direction::forward>() );
		    hipError_t error_code = hipSuccess;
		    auto workspace = make_workspace<FFT>(error_code);
			int shared_mem = sizeof(FFT::value_type)*(output_image.dims.y) + FFT::shared_memory_size*8;
			block_fft_kernel_C2C_rotate<FFT><< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> >
			( (typename FFT::output_type*)d_rotated_buffer,(typename FFT::output_type*)d_rotated_buffer, input_image.dims, output_image.dims, workspace);


		}
		else
		{
		    using FFT = decltype(FFT_4096_c2c() + Direction<fft_direction::inverse>() );
		    hipError_t error_code = hipSuccess;
		    auto workspace = make_workspace<FFT>(error_code);
			int shared_mem = sizeof(FFT::value_type)*(output_image.dims.y) + FFT::shared_memory_size*8;
			block_fft_kernel_C2C_rotate<FFT><< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> >
			( (typename FFT::output_type*)d_rotated_buffer,(typename FFT::output_type*)d_rotated_buffer, input_image.dims, output_image.dims, workspace);


		}



	}
	else
	{
		exit(-1);
	}




}

template<class FFT>
__global__ void block_fft_kernel_C2C_rotate(typename FFT::input_type* input_values, typename FFT::output_type* output_values, int4 dims_in, int4 dims_out, typename FFT::workspace_type workspace)
{

//	// Initialize the shared memory, assuming everyting matches the input data X size in
    using complex_type = typename FFT::value_type;
    using scalar_type  = typename complex_type::value_type;

	extern __shared__  complex_type shared_mem[];

    complex_type thread_data[FFT::storage_size];
    int rotated_offset = blockIdx.x * dims_out.y;

    bah_io::io<FFT>::load(&input_values[rotated_offset], thread_data, 1);


    // For loop zero the twiddles don't need to be computed
    FFT().execute(thread_data, shared_mem);

    bah_io::io<FFT>::load(thread_data, &output_values[rotated_offset], 1);



	return;

}

void DFTbyDecomposition::FFT_C2R_rotate()
{

	// This is the first set of 1d ffts when the input data are real valued, accessing the strided dimension. Since we need the full length, it will actually run a C2C xform

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");

	// Elements per thread must be [2,32]
    const int ept = 8;

	int threadsPerBlock = 4096/8; // FIXME make sure its a multiple of 32
	int gridDims = output_image.dims.w/2;


	if (input_image.dims.y == 4096)
	{
	    using FFT = decltype(FFT_4096_c2r() + Direction<fft_direction::inverse>() );
		int shared_mem = sizeof(FFT::value_type)*(input_image.dims.x) + FFT::shared_memory_size*8;

	    hipError_t error_code = hipSuccess;
	    auto workspace = make_workspace<FFT>(error_code);
		block_fft_kernel_C2R_rotate<FFT><< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( (typename FFT::output_type*)d_rotated_buffer, (float *)input_image.real_values_gpu, input_image.dims, output_image.dims, workspace);

	}
	else
	{
		exit(-1);
	}




}

template<class FFT>
__global__ void block_fft_kernel_C2R_rotate(typename FFT::input_type* input_values, float* output_values, int4 dims_in, int4 dims_out, typename FFT::workspace_type workspace)
{

//	// Initialize the shared memory, assuming everyting matches the input data X size in
	//	// Initialize the shared memory, assuming everyting matches the input data X size in
	using complex_type = typename FFT::value_type;
	using scalar_type  = typename complex_type::value_type;

	extern __shared__  complex_type shared_mem[];

	complex_type thread_data[FFT::storage_size];
	int source_idx[FFT::storage_size];

	bah_io::io<FFT>::load(&input_values[blockIdx.x * dims_out.y], thread_data, 1);


    // For loop zero the twiddles don't need to be computed
    FFT().execute(thread_data, shared_mem);

    int rotated_offset[2] = { -dims_out.w,  (dims_out.w/2 - blockIdx.x - 1)};

    bah_io::io<FFT>::store_c2r_rotated(thread_data, output_values, 1, rotated_offset);



	return;

}

