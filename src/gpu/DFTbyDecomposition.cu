#include "hip/hip_runtime.h"
/*
 * DFTbyDecomposition.cpp
 *
 *  Created on: Oct 21, 2020
 *      Author: himesb
 */

#include "gpu_core_headers.h"

DFTbyDecomposition::DFTbyDecomposition() // @suppress("Class members should be properly initialized")
{

}

DFTbyDecomposition::~DFTbyDecomposition()
{

}

DFTbyDecomposition::DFTbyDecomposition(const DFTbyDecomposition &other)
{
	// TODO Auto-generated constructor stub

}

DFTbyDecomposition& DFTbyDecomposition::operator=(
		const DFTbyDecomposition &other) {
	// TODO Auto-generated method stub

}

void DFTbyDecomposition::InitTestCase(int wanted_input_size_x, int wanted_input_size_y, int wanted_output_size_x, int wanted_output_size_y)
{
	dims_input = make_int2(wanted_input_size_x, wanted_input_size_y);
	dims_output = make_int2(wanted_output_size_x, wanted_output_size_y);

	// In practice we'll give a pointer to the arrays in some GpuImages
}

void DFTbyDecomposition::SetGpuImages(Image& cpu_input, Image& cpu_output)
{

	// Should be in real space, TODO add check
	input_image.CopyFromCpuImage(cpu_input);
	input_image.CopyHostToDevice();

	// Initialize to Fourier space
	output_image.CopyFromCpuImage(cpu_output);
	output_image.Allocate((int)dims_output.x, (int)dims_output.y, 1, false);

}
