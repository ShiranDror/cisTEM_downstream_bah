#include "hip/hip_runtime.h"
/*
 * DFTbyDecomposition.cpp
 *
 *  Created on: Oct 21, 2020
 *      Author: himesb
 */

#include "gpu_core_headers.h"
#include "/groups/himesb/cufftdx/include/cufftdx.hpp"

__global__ void DFT_R2C_WithPaddingKernel(hipfftReal* input_values, hipfftComplex* output_values, int4 dims_in, int4 dims_out, float C);
__global__ void DFT_C2C_WithPaddingKernel_strided(hipfftComplex* input_values, int4 dims_in, int4 dims_out, float C);
__global__ void DFT_R2C_WithPaddingKernel_strided(hipfftReal* input_values, hipfftComplex* output_values, int4 dims_in, int4 dims_out, float C);
__global__ void DFT_C2C_WithPaddingKernel(hipfftComplex* input_values, int4 dims_in, int4 dims_out, float C);
__global__ void DFT_C2C_WithPaddingKernel_rdx2(hipfftComplex* input_values, int4 dims_in, int4 dims_out, float C);

template<class FFT>
__global__ void block_fft_kernel_R2C_strided(hipfftReal* input_values, hipfftComplex* output_values, int4 dims_in, int4 dims_out, float CN, float CQ, int IQ);

DFTbyDecomposition::DFTbyDecomposition() // @suppress("Class members should be properly initialized")
{
	is_set_gpu_images = false;
	is_set_twiddles = false;
//	is_set_outputs = false;
}

DFTbyDecomposition::~DFTbyDecomposition()
{
	if (is_set_twiddles)
	{
		cudaErr(hipFree(twiddles));
	}
//	if (is_set_outputs)
//	{
//		cudaErr(hipFree(output_real));
//		cudaErr(hipFree(output_imag));
//	}
}

DFTbyDecomposition::DFTbyDecomposition(const DFTbyDecomposition &other)
{
	// TODO Auto-generated constructor stub

}

DFTbyDecomposition& DFTbyDecomposition::operator=(
		const DFTbyDecomposition &other) {
	// TODO Auto-generated method stub

}

void DFTbyDecomposition::InitTestCase(int wanted_input_size_x, int wanted_input_size_y, int wanted_output_size_x, int wanted_output_size_y)
{
	dims_input = make_int2(wanted_input_size_x, wanted_input_size_y);
	dims_output = make_int2(wanted_output_size_x, wanted_output_size_y);

	// In practice we'll give a pointer to the arrays in some GpuImages
}

void DFTbyDecomposition::SetGpuImages(Image& cpu_input, Image& cpu_output)
{

	// Should be in real space, TODO add check
	input_image.CopyFromCpuImage(cpu_input);
	input_image.CopyHostToDevice();


	// Initialize to Fourier space
	output_image.CopyFromCpuImage(cpu_output);
	output_image.Allocate((int)dims_output.x, (int)dims_output.y, 1, false);
	output_image.Zeros();



	is_set_gpu_images = true;

}


void DFTbyDecomposition::DFT_R2C_WithPadding()
{

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");



	int threadsPerBlock = input_image.dims.x; // FIXME make sure its a multiple of 32
	int gridDims = input_image.dims.y;
//	dim3 gridDims = dim3((output_image.dims.w/2 + threadsPerBlock - 1) / threadsPerBlock,
//					  	1, 1);
//  output_image.dims.y
	int shared_mem = sizeof(float)*input_image.dims.x;
	float C = -2*PIf/output_image.dims.x;
	DFT_R2C_WithPaddingKernel<< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( input_image.real_values_gpu,  output_image.complex_values_gpu, input_image.dims, output_image.dims, C);
	hipStreamSynchronize(hipStreamPerThread);



}

__global__ void DFT_R2C_WithPaddingKernel(hipfftReal* input_values, hipfftComplex* output_values, int4 dims_in, int4 dims_out, float C)
{

//	// Initialize the shared memory, assuming everying matches the input data X size in
	extern __shared__ float s[];
	// Avoid N*k type conversion and multiplication
	float* data = s;
//	float* coeff= (float*)&data[dims_in.x];


	int x = threadIdx.x;
	int pixel_out = (dims_out.w/2)*blockIdx.x;


	data[x] = __ldg((const float *)&input_values[dims_in.w*blockIdx.x + x]);
	__syncthreads();
//
//	 Loop over N updating the actual twiddle value along the way. This might lead to accuracy problems.
	float sum_real;
	float sum_imag;
	float twi_r;
	float twi_i;
	float coeff;

	for (int k = threadIdx.x; k < dims_out.w/2; k+=blockDim.x)
	{
		coeff = C*(float)k;
		sum_real = 0.0f;
		sum_imag = 0.0f;
		for (int n = 0; n < dims_in.x; n++)
		{
			__sincosf(coeff*n,&twi_i,&twi_r);
			sum_real = __fmaf_rn(data[n],twi_r,sum_real);
			sum_imag = __fmaf_rn(data[n],twi_i,sum_imag);
		}

		// Not sure if an async write, or storage to a shared mem temp would be faster.
		output_values[pixel_out + k].x = sum_real;
		output_values[pixel_out + k].y = sum_imag;
	}


	return;

}


void DFTbyDecomposition::DFT_C2C_WithPadding()
{

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");


	int threadsPerBlock = input_image.dims.x; // FIXME make sure its a multiple of 32
	int gridDims = output_image.dims.w/2;

	int shared_mem = sizeof(hipfftComplex)*input_image.dims.x;

	float C = -2*PIf/output_image.dims.x;
	DFT_C2C_WithPaddingKernel<< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( output_image.complex_values_gpu, input_image.dims, output_image.dims, C);
	hipStreamSynchronize(hipStreamPerThread);



}

__global__ void DFT_C2C_WithPaddingKernel(hipfftComplex* inplace_image, int4 dims_in, int4 dims_out, float C)
{

	// Initialize the shared memory, assuming everying matches the input data X size in
	// Check that setting hipFuncSetSharedMemConfig  to 8byte makes any diff for complex reads
	extern __shared__ hipfftComplex c[];
	hipfftComplex* data = c;


	int x = threadIdx.x;
	int pixel_out = (dims_out.w/2)*blockIdx.x;

	data[x] = __ldg((const hipfftComplex *)&inplace_image[pixel_out + x]);
	__syncthreads();
//
//	 Loop over N updating the actual twiddle value along the way. This might lead to accuracy problems.
	hipfftComplex sum;
	float twi_r;
	float twi_i;
	float coeff;
	float tmp;

	for (int k = threadIdx.x; k < dims_out.w/2; k+=blockDim.x)
	{
		coeff = C*(float)k;
		sum.x = 0.0f;
		sum.y = 0.0f;
		for (int n = 0; n < dims_in.y; n++)
		{
			__sincosf(coeff*n,&twi_i,&twi_r);
			tmp = data[n].x * twi_i;
			sum.x += __fmaf_rn(data[n].x, twi_r, -twi_i * data[n].y);
			sum.y += __fmaf_rn(data[n].y, twi_r, tmp);
		}

		// Not sure if an async write, or storage to a shared mem temp would be faster.
//		inplace_image[pixel_out + k].x = sum_real;
//		inplace_image[pixel_out + k].y = sum_imag;
		inplace_image[pixel_out + k] = sum;
	}



	return;

}


void DFTbyDecomposition::DFT_R2C_WithPadding_strided()
{

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");



	int threadsPerBlock = input_image.dims.y; // FIXME make sure its a multiple of 32
	int gridDims = input_image.dims.x;
//	dim3 gridDims = dim3((output_image.dims.w/2 + threadsPerBlock - 1) / threadsPerBlock,
//					  	1, 1);
//  output_image.dims.y
	int shared_mem = sizeof(float)*input_image.dims.y;
	float C = -2*PIf/output_image.dims.y;
	DFT_R2C_WithPaddingKernel_strided<< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( input_image.real_values_gpu,  output_image.complex_values_gpu, input_image.dims, output_image.dims, C);
	hipStreamSynchronize(hipStreamPerThread);



}

__global__ void DFT_R2C_WithPaddingKernel_strided(hipfftReal* input_values, hipfftComplex* output_values, int4 dims_in, int4 dims_out, float C)
{

//	// Initialize the shared memory, assuming everying matches the input data X size in
	extern __shared__ float s[];
	// Avoid N*k type conversion and multiplication
	float* data = s;
//	float* coeff= (float*)&data[dims_in.x];


	int y = threadIdx.x;
	int pixel_in = blockIdx.x + y * (dims_in.w);

	data[y] = __ldg((const hipfftReal *)&input_values[pixel_in]);
	__syncthreads();
//

//
//	 Loop over N updating the actual twiddle value along the way. This might lead to accuracy problems.
	float sum_real;
	float sum_imag;
	float twi_r;
	float twi_i;
	float coeff;

	for (int k = threadIdx.x; k < dims_out.y; k+=blockDim.x)
	{
		coeff = C*(float)k;
		sum_real = 0.0f;
		sum_imag = 0.0f;
		for (int n = 0; n < dims_in.x; n++)
		{
			__sincosf(coeff*n,&twi_i,&twi_r);
			sum_real = __fmaf_rn(data[n],twi_r,sum_real);
			sum_imag = __fmaf_rn(data[n],twi_i,sum_imag);
		}

		// Not sure if an async write, or storage to a shared mem temp would be faster.
		// Not sure if an async write, or storage to a shared mem temp would be faster.
		output_values[blockIdx.x + k * (dims_out.w/2)].x = sum_real;
		output_values[blockIdx.x + k * (dims_out.w/2)].y = sum_imag;
	}


	return;

}


void DFTbyDecomposition::DFT_C2C_WithPadding_strided()
{

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");


	int threadsPerBlock = input_image.dims.y; // FIXME make sure its a multiple of 32
	int gridDims = output_image.dims.w/2;

	int shared_mem = sizeof(hipfftComplex)*input_image.dims.y;

	float C = -2*PIf/output_image.dims.y;
	DFT_C2C_WithPaddingKernel_strided<< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( output_image.complex_values_gpu, input_image.dims, output_image.dims, C);
	hipStreamSynchronize(hipStreamPerThread);



}

__global__ void DFT_C2C_WithPaddingKernel_strided(hipfftComplex* inplace_image, int4 dims_in, int4 dims_out, float C)
{

	// Initialize the shared memory, assuming everying matches the input data X size in
	// Check that setting hipFuncSetSharedMemConfig  to 8byte makes any diff for complex reads
	extern __shared__ hipfftComplex c[];
	hipfftComplex* data = c;


	int y = threadIdx.x;
	int pixel_in = blockIdx.x + y * (dims_out.w/2);


	data[y] = __ldg((const hipfftComplex *)&inplace_image[pixel_in]);
	__syncthreads();
//
//	 Loop over N updating the actual twiddle value along the way. This might lead to accuracy problems.
	float sum_real;
	float sum_imag;
	float twi_r;
	float twi_i;
	float coeff;
	float tmp;

	for (int k = threadIdx.x; k < dims_out.y; k+=blockDim.x)
	{
		coeff = C*(float)k;
		sum_real = 0.0f;
		sum_imag = 0.0f;
		for (int n = 0; n < dims_in.y; n++)
		{
			__sincosf(coeff*n,&twi_i,&twi_r);
			tmp = data[n].x * twi_i;
			sum_real += __fmaf_rn(data[n].x, twi_r, -twi_i * data[n].y);
			sum_imag += __fmaf_rn(data[n].y, twi_r, tmp);
		}

		// Not sure if an async write, or storage to a shared mem temp would be faster.
		inplace_image[blockIdx.x + k * (dims_out.w/2)].x = sum_real;
		inplace_image[blockIdx.x + k * (dims_out.w/2)].y = sum_imag;
	}


	return;

}

void DFTbyDecomposition::DFT_C2C_WithPadding_rdx2()
{

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");


	int threadsPerBlock = input_image.dims.x; // FIXME make sure its a multiple of 32
	int gridDims = output_image.dims.w/2;

	int shared_mem = sizeof(hipfftComplex)*input_image.dims.x;

	float C = -2*PIf/output_image.dims.x*2;
	DFT_C2C_WithPaddingKernel_rdx2<< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( output_image.complex_values_gpu, input_image.dims, output_image.dims, C);
	hipStreamSynchronize(hipStreamPerThread);



}

__global__ void DFT_C2C_WithPaddingKernel_rdx2(hipfftComplex* inplace_image, int4 dims_in, int4 dims_out, float C)
{

	// Initialize the shared memory, assuming everying matches the input data X size in
	// Check that setting hipFuncSetSharedMemConfig  to 8byte makes any diff for complex reads
	extern __shared__ hipfftComplex c[];
	hipfftComplex* data = c;


	int x = threadIdx.x;
	int pixel_out = (dims_out.w/2)*blockIdx.x;

	data[x] = __ldg((const hipfftComplex *)&inplace_image[pixel_out + x]);
	__syncthreads();
//
//	 Loop over N updating the actual twiddle value along the way. This might lead to accuracy problems.
	hipfftComplex sum;
	hipfftComplex eve;
	float twi_r;
	float twi_i;
	float coeff;
	float tmp;

	for (int k = threadIdx.x; k < dims_out.w/4; k+=blockDim.x)
	{
		// get the even DFT
		coeff = C*(float)k;
		sum.x = 0.0f;
		sum.y = 0.0f;
		for (int n = 0; n < dims_in.y; n+=2)
		{
			__sincosf(coeff*n,&twi_i,&twi_r);
			tmp = data[n].x * twi_i;
			sum.x += __fmaf_rn(data[n].x, twi_r, -twi_i * data[n].y);
			sum.y += __fmaf_rn(data[n].y, twi_r, tmp);
		}

		eve = sum;

		// get the odd DFT
		sum.x = 0.0f;
		sum.y = 0.0f;
		for (int n = 1; n < dims_in.y; n+=2)
		{
			__sincosf(coeff*n,&twi_i,&twi_r);
			tmp = data[n].x * twi_i;
			sum.x += __fmaf_rn(data[n].x, twi_r, -twi_i * data[n].y);
			sum.y += __fmaf_rn(data[n].y, twi_r, tmp);
		}

		// Get the twiddle for the combined radix
		__sincosf(coeff/2.0f,&twi_i,&twi_r);
		// Multiply the odd
		tmp = sum.x * twi_i;
		sum.x = __fmaf_rn(sum.x, twi_r, -twi_i * sum.y);
		sum.y = __fmaf_rn(sum.y, twi_r, tmp);

		inplace_image[pixel_out + k].x = eve.x + sum.x;
		inplace_image[pixel_out + k].y = eve.y + sum.y;

		inplace_image[pixel_out + k + dims_out.w/4].x = eve.x - sum.x;
		inplace_image[pixel_out + k + dims_out.w/4].y = eve.y - sum.y;

	}



	return;

}


void DFTbyDecomposition::FFT_R2C_WithPadding_strided()
{

	// FIXME when adding real space complex images
	MyAssertTrue( input_image.is_in_memory_gpu, "Input image is in not on the GPU!");
	MyAssertTrue( output_image.is_in_memory_gpu, "Output image is in not on the GPU!");


    const int ept = 2;

	int threadsPerBlock = input_image.dims.y / ept; // FIXME make sure its a multiple of 32
	int gridDims = input_image.dims.x;
//	dim3 gridDims = dim3((output_image.dims.w/2 + threadsPerBlock - 1) / threadsPerBlock,
//					  	1, 1);
//  output_image.dims.y
	float CN = -2*PIf/output_image.dims.y;
	float CQ = -2*PIf/input_image.dims.y;
	int   IQ = output_image.dims.y / input_image.dims.y; // FIXME assuming for now this is already divisible
    using namespace cufftdx;

    // FFT is defined, its: size, type, direction, precision. Block() operator informs that FFT
    // will be executed on block level. Shared memory is required for co-operation between threads.
    using FFT          = decltype(Block() + Size<256>() + Type<fft_type::c2c>() + Direction<fft_direction::forward>() +
                         Precision<float>() + ElementsPerThread<ept>() + FFTsPerBlock<1>() + SM<700>());
//    using complex_type = typename FFT::value_type;
//    using real_type    = typename complex_type::value_type;

    using complex_type = typename FFT::value_type;

	int shared_mem = sizeof(float)*(2+input_image.dims.y) + FFT::shared_memory_size;
//	wxPrintf("IQ is %d %d %d\n",IQ,FFT::shared_memory_size, FFT::storage_size);


    // Invokes kernel with FFT::block_dim threads in CUDA block
	block_fft_kernel_R2C_strided<FFT><< <gridDims, threadsPerBlock, shared_mem, hipStreamPerThread>> > ( input_image.real_values_gpu,  output_image.complex_values_gpu, input_image.dims, output_image.dims, CN,CQ,IQ);


}

template<class FFT>
//__launch_bounds__(FFT::max_threads_per_block) __global__
__global__ void block_fft_kernel_R2C_strided(hipfftReal* input_values, hipfftComplex* output_values, int4 dims_in, int4 dims_out, float CN, float CQ, int IQ)
{

//	// Initialize the shared memory, assuming everying matches the input data X size in
    using complex_type = typename FFT::value_type;
    using scalar_type  = typename complex_type::value_type;
	extern __shared__  float real_data[];

	complex_type* results = (complex_type*)&real_data[dims_in.x];


	int y = threadIdx.x;
    real_data[y] = __ldg((const hipfftReal *)&input_values[blockIdx.x + y * (dims_in.w)]);
    real_data[y+128] = __ldg((const hipfftReal *)&input_values[blockIdx.x + (y+128) * (dims_in.w)]);

	__syncthreads();



	// Memory used by FFT
	complex_type twiddle;
    complex_type thread_data[2];

    float CN2 = CN * (y+128);
    CN*=128;
    int IQ2 = IQ * (y+128);
    IQ *= y;

    // For loop zero the twiddles don't need to be computed
    thread_data[1].x = real_data[y];
    thread_data[1].y = 0.0f;
    thread_data[2].x = real_data[y+128];
    thread_data[2].y = 0.0f;


    FFT().execute(thread_data, results);
    output_values[blockIdx.x + IQ * (dims_out.w/2)].x = (float)thread_data[1].x;
    output_values[blockIdx.x + IQ * (dims_out.w/2)].y = (float)thread_data[1].y;
    output_values[blockIdx.x + IQ2 * (dims_out.w/2)].x = (float)thread_data[2].x;
    output_values[blockIdx.x + IQ2 * (dims_out.w/2)].y = (float)thread_data[2].y;

    // For the other fragments we need the initial twiddle
	for (int fft_fragment = 0; fft_fragment < IQ; fft_fragment++)
	{
		// Pre shift with twiddle
		__sincosf(CN*fft_fragment,&twiddle.x,&twiddle.y);
	      thread_data[1].x = real_data[y] * twiddle.x;
	      thread_data[1].y = real_data[y] * twiddle.y;
		__sincosf(CN2*fft_fragment,&twiddle.x,&twiddle.y);
	      thread_data[2].x = real_data[y+128] * twiddle.x;
	      thread_data[2].y = real_data[y+128] * twiddle.y;
	      FFT().execute(thread_data, results);
//	      output_values[blockIdx.x + (fft_fragment + IQ) * (dims_out.w/2)] = thread_data;
//	      output_values[blockIdx.x + (fft_fragment + IQ) * (dims_out.w/2)].x = (float)thread_data.x;
	      output_values[blockIdx.x + (fft_fragment + IQ) * (dims_out.w/2)].x = (float)thread_data[1].x;
	      output_values[blockIdx.x + (fft_fragment + IQ) * (dims_out.w/2)].y = (float)thread_data[1].y;
	      output_values[blockIdx.x + (fft_fragment + IQ2) * (dims_out.w/2)].x = (float)thread_data[2].x;
	      output_values[blockIdx.x + (fft_fragment + IQ2) * (dims_out.w/2)].y = (float)thread_data[2].y;
	}



	return;

}
